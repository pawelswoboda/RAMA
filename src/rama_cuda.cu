#include <hip/hip_runtime.h>
#include "union_find.hxx"
#include "time_measure_util.h"
#include <algorithm>
#include <cstdlib>
#include "persistency_preprocessor.h"
#include "ECLgraph.h"
#include <thrust/transform_scan.h>
#include <thrust/transform.h>
#include "maximum_matching_vertex_based.h"
#include "maximum_matching_thrust.h"
#include "multicut_solver_options.h"
#include "dual_solver.h"
#include "edge_contractions_woc.h"
#include "rama_utils.h"

struct is_negative
{
    __host__ __device__
        bool operator()(const float x)
        {
            return x < 0.0;
        }
};
bool has_bad_contractions(const dCOO& A)
{
    const thrust::device_vector<float> d = A.diagonal();
    return thrust::count_if(d.begin(), d.end(), is_negative()) > 0;
}


struct map_nodes_to_new_clusters_func
{
    const int* node_mapping_cont_graph;
    int* node_mapping_orig_graph;
    const unsigned long num_nodes_cont;
    __host__ __device__ void operator()(const int n)
    {
        const int n_map = node_mapping_orig_graph[n];
        if (n_map < num_nodes_cont)
            node_mapping_orig_graph[n] = node_mapping_cont_graph[n_map];
    }
};


void map_node_labels(const thrust::device_vector<int>& cur_node_mapping, thrust::device_vector<int>& orig_node_mapping)
{
    map_nodes_to_new_clusters_func node_mapper({thrust::raw_pointer_cast(cur_node_mapping.data()), 
                                                thrust::raw_pointer_cast(orig_node_mapping.data()),
                                                cur_node_mapping.size()});

    thrust::for_each(thrust::make_counting_iterator<int>(0), thrust::make_counting_iterator<int>(0) + orig_node_mapping.size(), node_mapper);
}

std::tuple<thrust::device_vector<int>, int> contraction_mapping_by_maximum_matching(dCOO& A, const float mean_multiplier_mm, const bool verbose = true)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME;
    thrust::device_vector<int> node_mapping;
    int nr_matched_edges;
    std::tie(node_mapping, nr_matched_edges) = filter_edges_by_matching_vertex_based(A.export_undirected(), mean_multiplier_mm, verbose);
    // std::tie(node_mapping, nr_matched_edges) = filter_edges_by_matching_thrust(A, mean_multiplier_mm, verbose);
    return {compress_label_sequence(node_mapping, node_mapping.size() - 1), nr_matched_edges};
}

std::tuple<thrust::device_vector<int>, double, std::vector<std::vector<int>> > rama_cuda(dCOO& A, const multicut_solver_options& opts)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME;
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    thrust::device_vector<int> node_mapping(A.max_dim());
    thrust::sequence(node_mapping.begin(), node_mapping.end());

    if (opts.run_preprocessor) {
        auto[A_new, current_node_mapping] = preprocessor_cuda(A, opts,-1);
        map_node_labels(current_node_mapping,node_mapping);
        thrust::swap(A, A_new);
        if (opts.verbose)
            std::cout << "Energy after Preprocessor= " << A.sum() << "\n";
    }

    assert(A.is_directed());
    const double final_lb = dual_solver(A, opts.max_cycle_length_lb, opts.num_dual_itr_lb, opts.tri_memory_factor, opts.num_outer_itr_dual, 1e-4, opts.verbose);


    if (opts.verbose)
        std::cout << "initial energy = " << A.sum() << "\n";


    std::vector<std::vector<int>> timeline;

    if (opts.only_compute_lb)
        return {std::vector<int>(), final_lb, timeline};
        
    bool try_edges_to_contract_by_maximum_matching = true;
    if (opts.matching_thresh_crossover_ratio > 1.0)
        try_edges_to_contract_by_maximum_matching = false;


    for(size_t iter=0; A.nnz() > 0; ++iter)
    {
        if (iter > 0)
        {
            dual_solver(A, opts.max_cycle_length_primal, opts.num_dual_itr_primal, 1.0, 1, 1e-4, opts.verbose);
        }
        thrust::device_vector<int> cur_node_mapping;
        int nr_edges_to_contract;
        if(try_edges_to_contract_by_maximum_matching)
        {
            std::tie(cur_node_mapping, nr_edges_to_contract) = contraction_mapping_by_maximum_matching(A, opts.mean_multiplier_mm, opts.verbose);
            if(nr_edges_to_contract < A.rows() * opts.matching_thresh_crossover_ratio)
            {
                if (opts.verbose)
                {
                    std::cout << "# edges to contract = " << nr_edges_to_contract << ", # vertices = " << A.rows() << "\n";
                    std::cout << "switching to MST based contraction edge selection\n";
                }
                try_edges_to_contract_by_maximum_matching = false;    
            }
        }
        else
        {
            edge_contractions_woc c_mapper(A, opts.verbose);
            std::tie(cur_node_mapping, nr_edges_to_contract) = c_mapper.find_contraction_mapping();
        }


        if(nr_edges_to_contract == 0)
        {
            if (opts.verbose)
                std::cout << "# iterations = " << iter << "\n";
            break;
        }
        dCOO new_A = A.contract_cuda(cur_node_mapping);
        if (opts.verbose)
        {
            std::cout << "original A size " << A.cols() << "x" << A.rows() << "\n";
            std::cout << "contracted A size " << new_A.cols() << "x" << new_A.rows() << "\n";
        }
        assert(new_A.cols() < A.cols());

        if (opts.verbose)
        {
            const thrust::device_vector<float> diagonal = new_A.diagonal();
            const float energy_reduction = thrust::reduce(diagonal.begin(), diagonal.end());
            std::cout << "energy reduction " << energy_reduction << "\n";
        }
        if(has_bad_contractions(new_A))
            throw std::runtime_error("Found bad contractions");

        thrust::swap(A, new_A);
        A.remove_diagonal();
        if (opts.verbose)
            std::cout << "energy after iteration " << iter << ": " << A.sum() << ", #components = " << A.cols() << "\n";
        map_node_labels(cur_node_mapping, node_mapping);



         if (opts.run_preprocessor && opts.preprocessor_each_step) {
             auto [A_after_pp, pp_node_mapping] = preprocessor_cuda(A, opts,1);
             map_node_labels(pp_node_mapping,node_mapping);
             thrust::swap(A, A_after_pp);
         }


        if (opts.dump_timeline)
        {
            std::vector<int> current_timeline(node_mapping.size());
            thrust::copy(node_mapping.begin(), node_mapping.end(), current_timeline.begin());
            timeline.push_back(current_timeline);
        }
        if (opts.max_time_sec >= 0)
        {
            std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
            auto time = std::chrono::duration_cast<std::chrono::seconds>(end - begin).count();
            if (time > opts.max_time_sec)
                break;
        }
    }

    if (opts.verbose)
        std::cout << "final energy = " << A.sum() << "\n";



    return {node_mapping, final_lb, timeline};
}

std::tuple<std::vector<int>, double, int, std::vector<std::vector<int>> > rama_cuda(const std::vector<int>& i, const std::vector<int>& j, const std::vector<float>& costs, const multicut_solver_options& opts)
{
    initialize_gpu(opts.verbose);
    thrust::device_vector<int> i_gpu(i.begin(), i.end());
    thrust::device_vector<int> j_gpu(j.begin(), j.end());
    thrust::device_vector<float> costs_gpu(costs.begin(), costs.end());

    thrust::device_vector<int> sanitized_node_ids;
    if (opts.sanitize_graph)
        sanitized_node_ids = compute_sanitized_graph(i_gpu, j_gpu, costs_gpu);
    dCOO A(std::move(i_gpu), std::move(j_gpu), std::move(costs_gpu), true);


    double lb;
    std::vector<std::vector<int>> timeline;
    thrust::device_vector<int> node_mapping;
    std::chrono::steady_clock::time_point start_time = std::chrono::steady_clock::now();
    std::tie(node_mapping, lb, timeline) = rama_cuda(A, opts);

    std::chrono::steady_clock::time_point end_time = std::chrono::steady_clock::now();
    int time_duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();

    if (opts.sanitize_graph)
        node_mapping = desanitize_node_labels(node_mapping, sanitized_node_ids);
    std::vector<int> h_node_mapping(node_mapping.size());
    thrust::copy(node_mapping.begin(), node_mapping.end(), h_node_mapping.begin());
    return {h_node_mapping, lb, time_duration, timeline};
}

std::tuple<thrust::device_vector<int>, double, std::vector<std::vector<int>>> rama_cuda(thrust::device_vector<int>&& i, thrust::device_vector<int>&& j, thrust::device_vector<float>&& costs, const multicut_solver_options& opts, const int device)
{
    hipSetDevice(device);
    thrust::device_vector<int> sanitized_node_ids;
    if (opts.sanitize_graph)
        sanitized_node_ids = compute_sanitized_graph(i, j, costs);

    dCOO A(std::move(j), std::move(i), std::move(costs), true);
    thrust::device_vector<int> node_mapping;
    double lb;
    std::vector<std::vector<int>> timeline;
    
    std::tie(node_mapping, lb, timeline) = rama_cuda(A, opts);
    if (opts.sanitize_graph)
        node_mapping = desanitize_node_labels(node_mapping, sanitized_node_ids);

    return {node_mapping, lb, timeline};
}